#include "hip/hip_runtime.h"
#include "density_grid_renderer.cuh"
#include "kernel_launcher.hpp"
#include "utilities.hpp"

#include <hip/hip_runtime.h>
#include <>

#include <SFML/Graphics.hpp>

template<typename T>
__global__ void grid_to_image_kernel(element_accessor<T> const  grid,
                                     element_accessor<sf_pixel> image,
                                     size_t                     rows,
                                     size_t                     cols,
                                     color_multipliers          multipliers)
{
  for(size_t i = blockDim.y * blockIdx.y + threadIdx.y; i < rows; i += blockDim.y * gridDim.y)
  {
    for(size_t j = blockDim.x * blockIdx.x + threadIdx.x; j < cols; j += blockDim.x * gridDim.x)
    {
      const T&  grid_value = grid.at(j, i);
      sf_pixel& image_value = image.at(j, i);

      float clamped_value = max(0.0f, min(1.0f, grid_value));

      image_value.R = static_cast<uint8_t>(max(0.f, min(255.f, multipliers.r * grid_value)));
      image_value.G = static_cast<uint8_t>(max(0.f, min(255.f, multipliers.g * grid_value)));
      image_value.B = static_cast<uint8_t>(max(0.f, min(255.f, multipliers.b * grid_value)));
      image_value.A = 255U;
    }
  }
}

density_grid_renderer::density_grid_renderer(size_t const _rows, size_t const _cols)
  : grid_renderer(_rows, _cols)
  , m_grid_buffer{ _cols, _rows }
  , m_image_buffer{ _cols, _rows }
{
  m_texture.create(static_cast<unsigned int>(_cols), static_cast<unsigned int>(_rows));
  m_image.resize(_rows * _cols);
}

density_grid_renderer::~density_grid_renderer()
{}

void density_grid_renderer::draw(sf::RenderTarget& target, grid<float> const& grid, color_multipliers const& multipliers)
{
  auto error = copy(m_grid_buffer, grid);

  kernel_launcher::launch_2d(&grid_to_image_kernel<float>, cols(), rows(),
                             m_grid_buffer.accessor(), m_image_buffer.accessor(), rows(), cols(), multipliers);
  error = hipDeviceSynchronize();

  // 
  error = copy(m_image.data(), cols(), rows(), m_image_buffer, hipMemcpyDeviceToHost);
  m_texture.update(reinterpret_cast<uint8_t*>(m_image.data()), static_cast<unsigned int>(cols()), static_cast<unsigned int>(rows()), 0U, 0U);

  sf::Sprite sprite{ m_texture };
  sprite.setScale(target.getSize().x / sprite.getLocalBounds().width, target.getSize().y / sprite.getLocalBounds().height);
  target.draw(sprite);
}